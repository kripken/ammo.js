#include "hip/hip_runtime.h"
/*
Impulse based Rigid body simulation using CUDA
Copyright (c) 2007 Takahiro Harada  http://www.iii.u-tokyo.ac.jp/~takahiroharada/projects/impulseCUDA.html

This software is provided 'as-is', without any express or implied warranty.
In no event will the authors be held liable for any damages arising from the use of this software.
Permission is granted to anyone to use this software for any purpose, 
including commercial applications, and to alter it and redistribute it freely, 
subject to the following restrictions:

1. The origin of this software must not be misrepresented; you must not claim that you wrote the original software. If you use this software in a product, an acknowledgment in the product documentation would be appreciated but is not required.
2. Altered source versions must be plainly marked as such, and must not be misrepresented as being the original software.
3. This notice may not be removed or altered from any source distribution.
*/

#include <cstdlib>
#include <cstdio>
#include <string.h>

#include "cutil_math.h"
#include "hip/hip_math_constants.h"

#include <hip/hip_vector_types.h>



#include "btCudaDefines.h"



#include "../../src/BulletMultiThreaded/btGpuUtilsSharedDefs.h"
#include "../../Demos/Gpu2dDemo/btGpuDemo2dSharedTypes.h"
#include "../../Demos/Gpu2dDemo/btGpuDemo2dSharedDefs.h"



texture<float4, 1, hipReadModeElementType> posTex;



#include "../../Demos/Gpu2dDemo/btGpuDemo2dSharedCode.h"

