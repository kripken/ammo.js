#include "hip/hip_runtime.h"
/*
Bullet Continuous Collision Detection and Physics Library, http://bulletphysics.org
Copyright (C) 2006, 2007 Sony Computer Entertainment Inc. 

This software is provided 'as-is', without any express or implied warranty.
In no event will the authors be held liable for any damages arising from the use of this software.
Permission is granted to anyone to use this software for any purpose, 
including commercial applications, and to alter it and redistribute it freely, 
subject to the following restrictions:

1. The origin of this software must not be misrepresented; you must not claim that you wrote the original software. If you use this software in a product, an acknowledgment in the product documentation would be appreciated but is not required.
2. Altered source versions must be plainly marked as such, and must not be misrepresented as being the original software.
3. This notice may not be removed or altered from any source distribution.
*/

#include <cstdlib>
#include <cstdio>
#include <string.h>

#include "cutil_math.h"
#include "hip/hip_math_constants.h"


#include <hip/hip_vector_types.h>



#include "btCudaDefines.h"



#include "../../src/BulletMultiThreaded/btGpuUtilsSharedDefs.h"
#include "../../src/BulletMultiThreaded/btGpu3DGridBroadphaseSharedDefs.h"



__device__ inline bt3DGrid3F1U tex_fetch3F1U(float4 a) { return *((bt3DGrid3F1U*)(&a)); }



void btCuda_exit(int val);



texture<uint2, 1, hipReadModeElementType> particleHashTex;
texture<uint, 1, hipReadModeElementType> cellStartTex;
texture<float4, 1, hipReadModeElementType> pAABBTex;



__constant__ bt3DGridBroadphaseParams params;



extern "C"
{



void btCuda_setParameters(bt3DGridBroadphaseParams* hostParams)
{
    // copy parameters to constant memory
    BT_GPU_SAFE_CALL(hipMemcpyToSymbol(HIP_SYMBOL(params), hostParams, sizeof(bt3DGridBroadphaseParams)));
}



} // extern "C"



#include "../../src/BulletMultiThreaded/btGpu3DGridBroadphaseSharedCode.h"


