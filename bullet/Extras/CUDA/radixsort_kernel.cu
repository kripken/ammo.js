#include "hip/hip_runtime.h"
/*
 * Copyright 1993-2006 NVIDIA Corporation.  All rights reserved.
 *
 * NOTICE TO USER:   
 *
 * This source code is subject to NVIDIA ownership rights under U.S. and 
 * international Copyright laws.  
 *
 * NVIDIA MAKES NO REPRESENTATION ABOUT THE SUITABILITY OF THIS SOURCE 
 * CODE FOR ANY PURPOSE.  IT IS PROVIDED "AS IS" WITHOUT EXPRESS OR 
 * IMPLIED WARRANTY OF ANY KIND.  NVIDIA DISCLAIMS ALL WARRANTIES WITH 
 * REGARD TO THIS SOURCE CODE, INCLUDING ALL IMPLIED WARRANTIES OF 
 * MERCHANTABILITY, NONINFRINGEMENT, AND FITNESS FOR A PARTICULAR PURPOSE.   
 * IN NO EVENT SHALL NVIDIA BE LIABLE FOR ANY SPECIAL, INDIRECT, INCIDENTAL, 
 * OR CONSEQUENTIAL DAMAGES, OR ANY DAMAGES WHATSOEVER RESULTING FROM LOSS 
 * OF USE, DATA OR PROFITS, WHETHER IN AN ACTION OF CONTRACT, NEGLIGENCE 
 * OR OTHER TORTIOUS ACTION, ARISING OUT OF OR IN CONNECTION WITH THE USE 
 * OR PERFORMANCE OF THIS SOURCE CODE.  
 *
 * U.S. Government End Users.  This source code is a "commercial item" as 
 * that term is defined at 48 C.F.R. 2.101 (OCT 1995), consisting  of 
 * "commercial computer software" and "commercial computer software 
 * documentation" as such terms are used in 48 C.F.R. 12.212 (SEPT 1995) 
 * and is provided to the U.S. Government only as a commercial end item.  
 * Consistent with 48 C.F.R.12.212 and 48 C.F.R. 227.7202-1 through 
 * 227.7202-4 (JUNE 1995), all U.S. Government End Users acquire the 
 * source code with only those rights set forth herein.
 */

/* Radixsort project with key/value and arbitrary datset size support
 * which demonstrates the use of CUDA in a multi phase sorting 
 * computation.
 * Device code.
 */

#ifndef _RADIXSORT_KERNEL_H_
#define _RADIXSORT_KERNEL_H_

#include <stdio.h>
#include "radixsort.cuh"

#define SYNCIT __syncthreads()

static const int NUM_SMS = 16;
static const int NUM_THREADS_PER_SM = 192;
static const int NUM_THREADS_PER_BLOCK = 64;
//static const int NUM_THREADS = NUM_THREADS_PER_SM * NUM_SMS;
static const int NUM_BLOCKS = (NUM_THREADS_PER_SM / NUM_THREADS_PER_BLOCK) * NUM_SMS;
static const int RADIX = 8;                                                        // Number of bits per radix sort pass
static const int RADICES = 1 << RADIX;                                             // Number of radices
static const int RADIXMASK = RADICES - 1;                                          // Mask for each radix sort pass
#if SIXTEEN
static const int RADIXBITS = 16;                                                   // Number of bits to sort over
#else
static const int RADIXBITS = 32;                                                   // Number of bits to sort over
#endif
static const int RADIXTHREADS = 16;                                                // Number of threads sharing each radix counter
static const int RADIXGROUPS = NUM_THREADS_PER_BLOCK / RADIXTHREADS;               // Number of radix groups per CTA
static const int TOTALRADIXGROUPS = NUM_BLOCKS * RADIXGROUPS;                      // Number of radix groups for each radix
static const int SORTRADIXGROUPS = TOTALRADIXGROUPS * RADICES;                     // Total radix count
static const int GRFELEMENTS = (NUM_THREADS_PER_BLOCK / RADIXTHREADS) * RADICES; 
static const int GRFSIZE = GRFELEMENTS * sizeof(uint); 

// Prefix sum variables
static const int PREFIX_NUM_THREADS_PER_SM = NUM_THREADS_PER_SM;
static const int PREFIX_NUM_THREADS_PER_BLOCK = PREFIX_NUM_THREADS_PER_SM;
static const int PREFIX_NUM_BLOCKS = (PREFIX_NUM_THREADS_PER_SM / PREFIX_NUM_THREADS_PER_BLOCK) * NUM_SMS;
static const int PREFIX_BLOCKSIZE = SORTRADIXGROUPS / PREFIX_NUM_BLOCKS;
static const int PREFIX_GRFELEMENTS = PREFIX_BLOCKSIZE + 2 * PREFIX_NUM_THREADS_PER_BLOCK;
static const int PREFIX_GRFSIZE = PREFIX_GRFELEMENTS * sizeof(uint);

// Shuffle variables
static const int SHUFFLE_GRFOFFSET = RADIXGROUPS * RADICES;
static const int SHUFFLE_GRFELEMENTS = SHUFFLE_GRFOFFSET + PREFIX_NUM_BLOCKS; 
static const int SHUFFLE_GRFSIZE = SHUFFLE_GRFELEMENTS * sizeof(uint); 


#define SDATA( index)      CUT_BANK_CHECKER(sdata, index)

// Prefix sum data
uint gRadixSum[TOTALRADIXGROUPS * RADICES];
__device__ uint dRadixSum[TOTALRADIXGROUPS * RADICES];
uint gRadixBlockSum[PREFIX_NUM_BLOCKS];
__device__ uint dRadixBlockSum[PREFIX_NUM_BLOCKS];

extern __shared__ uint sRadixSum[];



////////////////////////////////////////////////////////////////////////////////
//! Perform a radix sum on the list to be sorted.  Each SM holds a set of
//! radix counters for each group of RADIXGROUPS thread in the GRF. 
//!
//! @param pData     input data
//! @param elements  total number of elements
//! @param elements_rounded_to_3072  total number of elements rounded up to the 
//!                                  nearest multiple of 3072
//! @param shift     the shift (0 to 24) that we are using to obtain the correct 
//!                  byte
////////////////////////////////////////////////////////////////////////////////
__global__ void RadixSum(KeyValuePair *pData, uint elements, uint elements_rounded_to_3072, uint shift)
{
    uint pos    = threadIdx.x;

    // Zero radix counts
    while (pos < GRFELEMENTS)
    {
        sRadixSum[pos] = 0;
        pos += NUM_THREADS_PER_BLOCK;
    }

    // Sum up data
    // Source addresses computed so that each thread is reading from a block of 
    // consecutive addresses so there are no conflicts between threads
    // They then loop over their combined region and the next batch works elsewhere.
    // So threads 0 to 16 work on memory 0 to 320.
    // First reading 0,1,2,3...15 then 16,17,18,19...31 and so on
    // optimising parallel access to shared memory by a thread accessing 16*threadID
    // The next radix group runs from 320 to 640 and the same applies in that region
    uint tmod   =   threadIdx.x % RADIXTHREADS;
    uint tpos   =   threadIdx.x / RADIXTHREADS;

    // Take the rounded element list size so that all threads have a certain size dataset to work with
    // and no zero size datasets confusing the issue
    // By using a multiple of 3072 we ensure that all threads have elements
    // to work with until the last phase, at which point we individually test
    uint element_fraction  =   elements_rounded_to_3072 / TOTALRADIXGROUPS;

    // Generate range 
    // Note that it is possible for both pos and end to be past the end of the element set
    // which will be caught later.
    pos       = (blockIdx.x * RADIXGROUPS + tpos) * element_fraction;
    uint end  = pos + element_fraction;
    pos      += tmod; 
    //printf("pos: %d\n", pos);
    __syncthreads();

    while (pos < end )
    {
        uint key = 0;

        // Read first data element if we are in the set of elements
        //if( pos < elements )
            //key = pData[pos].key;
        KeyValuePair kvp;
        // Read first data element, both items at once as the memory will want to coalesce like that anyway
        if (pos < elements)
            kvp = pData[pos];
        else
            kvp.key = 0;
        key = kvp.key;


        // Calculate position of radix counter to increment
        // There are RADICES radices in each pass (256)
        // and hence this many counters for bin grouping
        // Multiply by RADIXGROUPS (4) to spread through memory
        // and into 4 radix groups
        uint p = ((key >> shift) & RADIXMASK) * RADIXGROUPS;
       
        // Increment radix counters
        // Each radix group has its own set of counters
        // so we add the thread position [0-3], ie the group index.
        // We slow down here and take at least 16 cycles to write to the summation boxes
        // but other groups will only conflict with themselves and so can also be writing
        // 16 cycles here at least avoids retries.
        uint ppos = p + tpos;

        // If we are past the last element we don't want to do anything
        // We do have to check each time, however, to ensure that all
        // threads sync on each sync here.
        if (tmod == 0 && pos < elements)
            sRadixSum[ppos]++;
            SYNCIT;
        if (tmod == 1 && pos < elements)
            sRadixSum[ppos]++;  
            SYNCIT;          
        if (tmod == 2 && pos < elements)
            sRadixSum[ppos]++;   
            SYNCIT; 
        if (tmod == 3 && pos < elements)
            sRadixSum[ppos]++; 
            SYNCIT;  
        if (tmod == 4 && pos < elements)
            sRadixSum[ppos]++;
            SYNCIT;
        if (tmod == 5 && pos < elements)
            sRadixSum[ppos]++; 
            SYNCIT;           
        if (tmod == 6 && pos < elements)
            sRadixSum[ppos]++;
            SYNCIT;   
        if (tmod == 7 && pos < elements)
            sRadixSum[ppos]++;  
            SYNCIT;   
        if (tmod == 8 && pos < elements)
            sRadixSum[ppos]++;
            SYNCIT;
        if (tmod == 9 && pos < elements)
            sRadixSum[ppos]++; 
            SYNCIT;           
        if (tmod == 10 && pos < elements)
            sRadixSum[ppos]++;
            SYNCIT;    
        if (tmod == 11 && pos < elements)
            sRadixSum[ppos]++; 
            SYNCIT;     
        if (tmod == 12 && pos < elements)
            sRadixSum[ppos]++;
            SYNCIT;
        if (tmod == 13 && pos < elements)
            sRadixSum[ppos]++; 
            SYNCIT;           
        if (tmod == 14 && pos < elements)
            sRadixSum[ppos]++; 
            SYNCIT;   
        if (tmod == 15 && pos < elements)
            sRadixSum[ppos]++; 
            SYNCIT;   
           
        pos += RADIXTHREADS;
                
    }

    __syncthreads();

    __syncthreads();

    // Output radix sums into separate memory regions for each radix group
    // So this memory then is layed out:
    // 0...... 192..... 384 ................ 192*256
    // ie all 256 bins for each radix group
    // in there:
    // 0.............192
    // 0  4  8  12...     - block idx * 4
    // And in the block boxes we see the 4 radix groups for that block
    // So 0-192 should contain bin 0 for each radix group, and so on
    uint offset = blockIdx.x * RADIXGROUPS;
    uint row    = threadIdx.x / RADIXGROUPS;
    uint column = threadIdx.x % RADIXGROUPS;
    while (row < RADICES)
    {
        dRadixSum[offset + row * TOTALRADIXGROUPS + column] = sRadixSum[row * RADIXGROUPS + column];
        row += NUM_THREADS_PER_BLOCK / RADIXGROUPS;
    }
}

////////////////////////////////////////////////////////////////////////////////
//! Performs first part of parallel prefix sum - individual sums of each radix
//! count. By the end of this we have prefix sums on a block level  in dRadixSum
//! and totals for blocks in dRadixBlockSum.
////////////////////////////////////////////////////////////////////////////////
__global__ void RadixPrefixSum()
{
    // Read radix groups in offset by one in the GRF so a zero can be inserted at the beginning
    // and the final sum of all radix counts summed here is tacked onto the end for reading by
    // the next stage
    // Each block in this case is the full number of threads per SM (and hence the total number 
    // of radix groups), 192. We should then have the total set of offsets for an entire radix 
    // group by the end of this stage
    // Device mem addressing
    
    uint brow       = blockIdx.x * (RADICES / PREFIX_NUM_BLOCKS);
    uint drow       = threadIdx.x / TOTALRADIXGROUPS; // In default parameterisation this is always 0
    uint dcolumn    = threadIdx.x % TOTALRADIXGROUPS; // And similarly this is always the same as threadIdx.x   
    uint dpos       = (brow + drow) * TOTALRADIXGROUPS + dcolumn;
    uint end        = ((blockIdx.x + 1) * (RADICES / PREFIX_NUM_BLOCKS)) * TOTALRADIXGROUPS;
    // Shared mem addressing
    uint srow       = threadIdx.x / (PREFIX_BLOCKSIZE / PREFIX_NUM_THREADS_PER_BLOCK);
    uint scolumn    = threadIdx.x % (PREFIX_BLOCKSIZE / PREFIX_NUM_THREADS_PER_BLOCK);
    uint spos       = srow * (PREFIX_BLOCKSIZE / PREFIX_NUM_THREADS_PER_BLOCK + 1) + scolumn;

    // Read (RADICES / PREFIX_NUM_BLOCKS) radix counts into the GRF alongside each other
    while (dpos < end)
    {
        sRadixSum[spos] = dRadixSum[dpos];
        spos += (PREFIX_NUM_THREADS_PER_BLOCK / (PREFIX_BLOCKSIZE / PREFIX_NUM_THREADS_PER_BLOCK)) * 
                (PREFIX_BLOCKSIZE / PREFIX_NUM_THREADS_PER_BLOCK + 1);
        dpos += (TOTALRADIXGROUPS / PREFIX_NUM_THREADS_PER_BLOCK) * TOTALRADIXGROUPS;
    }
    __syncthreads();
       
    // Perform preliminary sum on each thread's stretch of data
    // Each thread having a block of 16, with spacers between 0...16 18...33 and so on
    int pos     = threadIdx.x * (PREFIX_BLOCKSIZE / PREFIX_NUM_THREADS_PER_BLOCK + 1);
    end         = pos + (PREFIX_BLOCKSIZE / PREFIX_NUM_THREADS_PER_BLOCK);
    uint sum    = 0;
    while (pos < end)
    {
        sum += sRadixSum[pos];
        sRadixSum[pos] = sum;
        pos++;  
    }
    __syncthreads();   
  
 
    // Calculate internal offsets by performing a more traditional parallel
    // prefix sum of the topmost member of each thread's work data.  Right now,
    // these are stored between the work data for each thread, allowing us to 
    // eliminate GRF conflicts as well as hold the offsets needed to complete the sum
    // In other words we have:
    // 0....15 16 17....32 33 34....
    // Where this first stage updates the intermediate values (so 16=15, 33=32 etc)
    int m           = (PREFIX_BLOCKSIZE / PREFIX_NUM_THREADS_PER_BLOCK + 1);
    pos             = threadIdx.x  * (PREFIX_BLOCKSIZE / PREFIX_NUM_THREADS_PER_BLOCK + 1) +
                      (PREFIX_BLOCKSIZE / PREFIX_NUM_THREADS_PER_BLOCK);
    sRadixSum[pos]  = sRadixSum[pos - 1];
    __syncthreads();
    // This stage then performs a parallel prefix sum (ie use powers of 2 to propagate in log n stages)
    // to update 17, 34 etc with the totals to that point (so 34 becomes [34] + [17]) and so on.
    while (m < PREFIX_NUM_THREADS_PER_BLOCK * (PREFIX_BLOCKSIZE / PREFIX_NUM_THREADS_PER_BLOCK + 1))
    {
        int p  = pos - m;
        uint t = ((p > 0) ? sRadixSum[p] : 0);
        __syncthreads();
        sRadixSum[pos] += t;
        __syncthreads();
        m *= 2;
    } 
    __syncthreads();

  
  
    // Add internal offsets to each thread's work data.
    // So now we take 17 and add it to all values 18 to 33 so all offsets for that block 
    // are updated.
    pos     = threadIdx.x * (PREFIX_BLOCKSIZE / PREFIX_NUM_THREADS_PER_BLOCK + 1);
    end     = pos + (PREFIX_BLOCKSIZE / PREFIX_NUM_THREADS_PER_BLOCK);
    int p   = pos - 1;
    sum     = ((p > 0) ? sRadixSum[p] : 0);
    while (pos < end)
    {
        sRadixSum[pos] += sum;
        pos++; 
    }
    __syncthreads();        
               
    // Write summed data back out to global memory in the same way as we read it in
    // We now have prefix sum values internal to groups
    brow       = blockIdx.x * (RADICES / PREFIX_NUM_BLOCKS);
    drow       = threadIdx.x / TOTALRADIXGROUPS;
    dcolumn    = threadIdx.x % TOTALRADIXGROUPS;    
    srow       = threadIdx.x / (PREFIX_BLOCKSIZE / PREFIX_NUM_THREADS_PER_BLOCK);
    scolumn    = threadIdx.x % (PREFIX_BLOCKSIZE / PREFIX_NUM_THREADS_PER_BLOCK);
    dpos       = (brow + drow) * TOTALRADIXGROUPS + dcolumn + 1;
    spos       = srow * (PREFIX_BLOCKSIZE / PREFIX_NUM_THREADS_PER_BLOCK + 1) + scolumn;
    end        = ((blockIdx.x + 1) * RADICES / PREFIX_NUM_BLOCKS) * TOTALRADIXGROUPS;
    while (dpos < end)
    {
        dRadixSum[dpos] = sRadixSum[spos];
        dpos += (TOTALRADIXGROUPS / PREFIX_NUM_THREADS_PER_BLOCK) * TOTALRADIXGROUPS;        
        spos += (PREFIX_NUM_THREADS_PER_BLOCK / (PREFIX_BLOCKSIZE / PREFIX_NUM_THREADS_PER_BLOCK)) * 
                (PREFIX_BLOCKSIZE / PREFIX_NUM_THREADS_PER_BLOCK + 1);
    }

    // Write last element to summation
    // Storing block sums in a separate array
    if (threadIdx.x == 0) {
        dRadixBlockSum[blockIdx.x] = sRadixSum[PREFIX_NUM_THREADS_PER_BLOCK * (PREFIX_BLOCKSIZE / PREFIX_NUM_THREADS_PER_BLOCK + 1) - 1];
        dRadixSum[blockIdx.x * PREFIX_BLOCKSIZE] = 0;
    }
}


////////////////////////////////////////////////////////////////////////////////
//! Initially perform prefix sum of block totals to obtain final set of offsets.
//! Then make use of radix sums to perform a shuffling of the data into the 
//! correct bins.
//!
//! @param pSrc      input data
//! @param pDst      output data
//! @param elements  total number of elements
//! @param shift     the shift (0 to 24) that we are using to obtain the correct 
//!                  byte
////////////////////////////////////////////////////////////////////////////////
__global__ void RadixAddOffsetsAndShuffle(KeyValuePair* pSrc, KeyValuePair* pDst, uint elements, uint elements_rounded_to_3072, int shift)
{
    // Read offsets from previous blocks
    if (threadIdx.x == 0)
        sRadixSum[SHUFFLE_GRFOFFSET] = 0;
        
    if (threadIdx.x < PREFIX_NUM_BLOCKS - 1)
        sRadixSum[SHUFFLE_GRFOFFSET + threadIdx.x + 1] = dRadixBlockSum[threadIdx.x];
    __syncthreads();
    
    // Parallel prefix sum over block sums
    int pos = threadIdx.x;
    int n = 1;
    while (n < PREFIX_NUM_BLOCKS)
    {
        int ppos = pos - n;
        uint t0 = ((pos < PREFIX_NUM_BLOCKS) && (ppos >= 0)) ? sRadixSum[SHUFFLE_GRFOFFSET + ppos] : 0;
        __syncthreads();
        if (pos < PREFIX_NUM_BLOCKS)
            sRadixSum[SHUFFLE_GRFOFFSET + pos] += t0;
        __syncthreads(); 
        n *= 2;
    }

    // Read radix count data and add appropriate block offset
    // for each radix at the memory location for this thread
    // (where the other threads in the block will be reading
    // as well, hence the large stride).
    // There is one counter box per radix group per radix 
    // per block (4*256*3)
    // We use 64 threads to read the 4 radix groups set of radices 
    // for the block. 
    int row    = threadIdx.x / RADIXGROUPS;
    int column = threadIdx.x % RADIXGROUPS;
    int spos   = row * RADIXGROUPS + column;
    int dpos   = row * TOTALRADIXGROUPS + column + blockIdx.x * RADIXGROUPS;
    while (spos < SHUFFLE_GRFOFFSET)
    {
        sRadixSum[spos] = dRadixSum[dpos] + sRadixSum[SHUFFLE_GRFOFFSET + dpos / (TOTALRADIXGROUPS * RADICES / PREFIX_NUM_BLOCKS)];
        spos += NUM_THREADS_PER_BLOCK;
        dpos += (NUM_THREADS_PER_BLOCK / RADIXGROUPS) * TOTALRADIXGROUPS;
    }
    __syncthreads();

    //int pos;
    // Shuffle data
    // Each of the subbins for a block should be filled via the counters, properly interleaved
    // Then, as we now iterate over each data value, we increment the subbins (each thread in the 
    // radix group in turn to avoid miss writes due to conflicts) and set locations correctly.
    uint element_fraction  =   elements_rounded_to_3072 / TOTALRADIXGROUPS;
    int tmod   =   threadIdx.x % RADIXTHREADS;
    int tpos   =   threadIdx.x / RADIXTHREADS;

    pos       = (blockIdx.x * RADIXGROUPS + tpos) * element_fraction;
    uint end  = pos + element_fraction; //(blockIdx.x * RADIXGROUPS + tpos + 1) * element_fraction;
    pos      += tmod; 

    __syncthreads();

    while (pos < end )
    {
        KeyValuePair kvp;
#if 1 // old load
        // Read first data element, both items at once as the memory will want to coalesce like that anyway
        if (pos < elements)
        {
            kvp = pSrc[pos];
        }
        else
            kvp.key = 0;

#else // casting to float2 to get it to combine loads
        int2 kvpf2;

        // Read first data element, both items at once as the memory will want to coalesce like that anyway
        if (pos < elements)
        {
  //          kvp = pSrc[pos];
            kvpf2 = ((int2*)pSrc)[pos];
           // printf("kvp: %f %f  kvpf2: %f %f\n", kvp.key, kvp.value, kvpf2.x, kvpf2.y);
        }
        else
            //kvp.key = 0;
            kvpf2.x = 0;
 
        kvp.key = kvpf2.x;
        kvp.value = kvpf2.y;
#endif  

        uint index;
                
        // Calculate position of radix counter to increment
        uint p = ((kvp.key >> shift) & RADIXMASK) * RADIXGROUPS;
                
        // Move data, keeping counts updated.
        // Increment radix counters, relying on hexadecathread
        // warp to prevent this code from stepping all over itself.
        uint ppos = p + tpos;
        if (tmod == 0 && pos < elements)
        {
            index = sRadixSum[ppos]++;
            pDst[index] = kvp;
        }
            SYNCIT;
        if (tmod == 1 && pos < elements)
        {
            index = sRadixSum[ppos]++;
            pDst[index] = kvp;
        }
            SYNCIT;     
        if (tmod == 2 && pos < elements)
        {
            index = sRadixSum[ppos]++;
            pDst[index] = kvp;
        }
            SYNCIT;
        if (tmod == 3 && pos < elements)
        {
            index = sRadixSum[ppos]++;
            pDst[index] = kvp;
        }
            SYNCIT;
        if (tmod == 4 && pos < elements)
        {
            index = sRadixSum[ppos]++;
            pDst[index] = kvp;
        }
            SYNCIT;
        if (tmod == 5 && pos < elements)
        {
            index = sRadixSum[ppos]++;
            pDst[index] = kvp;
        }
            SYNCIT; 
        if (tmod == 6 && pos < elements)
        {
            index = sRadixSum[ppos]++;
            pDst[index] = kvp;
        }
            SYNCIT;
        if (tmod == 7 && pos < elements)
        {
            index = sRadixSum[ppos]++;
            pDst[index] = kvp;
        }
            SYNCIT; 
        if (tmod == 8 && pos < elements)
        {
            index = sRadixSum[ppos]++;
            pDst[index] = kvp;
        }
            SYNCIT;
        if (tmod == 9 && pos < elements)
        {
            index = sRadixSum[ppos]++;
            pDst[index] = kvp;
        }
            SYNCIT;          
        if (tmod == 10 && pos < elements)
        {
            index = sRadixSum[ppos]++;
            pDst[index] = kvp;
        }
            SYNCIT;
        if (tmod == 11 && pos < elements)
        {
            index = sRadixSum[ppos]++;
            pDst[index] = kvp;
        }
            SYNCIT;
        if (tmod == 12 && pos < elements)
        {
            index = sRadixSum[ppos]++;
            pDst[index] = kvp;
        }
            SYNCIT;
        if (tmod == 13 && pos < elements)
        {
            index = sRadixSum[ppos]++;
            pDst[index] = kvp;
        }
            SYNCIT;           
        if (tmod == 14 && pos < elements)
        {
            index = sRadixSum[ppos]++;
            pDst[index] = kvp;
        }
            SYNCIT;
        if (tmod == 15 && pos < elements)
        {
            index = sRadixSum[ppos]++;
            pDst[index] = kvp;
        }
            SYNCIT;                         

        pos += RADIXTHREADS;
    }

    __syncthreads();
}

#endif // #ifndef _RADIXSORT_KERNEL_H_
