/*
Bullet Continuous Collision Detection and Physics Library, http://bulletphysics.org
Copyright (C) 2006, 2007 Sony Computer Entertainment Inc. 

This software is provided 'as-is', without any express or implied warranty.
In no event will the authors be held liable for any damages arising from the use of this software.
Permission is granted to anyone to use this software for any purpose, 
including commercial applications, and to alter it and redistribute it freely, 
subject to the following restrictions:

1. The origin of this software must not be misrepresented; you must not claim that you wrote the original software. If you use this software in a product, an acknowledgment in the product documentation would be appreciated but is not required.
2. Altered source versions must be plainly marked as such, and must not be misrepresented as being the original software.
3. This notice may not be removed or altered from any source distribution.
*/

#include <cstdlib>
#include <cstdio>
#include <string.h>

#include <GL/glut.h>
#include <cuda_gl_interop.h>

#include "cutil_math.h"
#include "hip/hip_math_constants.h"

#include <hip/hip_vector_types.h>




#include "btCudaDefines.h"
#include "../../src/BulletMultiThreaded/btGpuUtilsSharedDefs.h"


void btCuda_exit(int val)
{
    fprintf(stderr, "Press ENTER key to terminate the program\n");
    getchar();
	exit(val);
}

void btCuda_allocateArray(void** devPtr, unsigned int size)
{
    BT_GPU_SAFE_CALL(hipMalloc(devPtr, size));
}

void btCuda_freeArray(void* devPtr)
{
    BT_GPU_SAFE_CALL(hipFree(devPtr));
}

void btCuda_copyArrayFromDevice(void* host, const void* device, unsigned int size)
{   
    BT_GPU_SAFE_CALL(hipMemcpy(host, device, size, hipMemcpyDeviceToHost));
}

void btCuda_copyArrayToDevice(void* device, const void* host, unsigned int size)
{
    BT_GPU_SAFE_CALL(hipMemcpy((char*)device, host, size, hipMemcpyHostToDevice));
}


void btCuda_registerGLBufferObject(unsigned int vbo)
{
    BT_GPU_SAFE_CALL(cudaGLRegisterBufferObject(vbo));
}

void* btCuda_mapGLBufferObject(unsigned int vbo)
{
    void *ptr;
    BT_GPU_SAFE_CALL(cudaGLMapBufferObject(&ptr, vbo));
    return ptr;
}

void btCuda_unmapGLBufferObject(unsigned int vbo)
{
    BT_GPU_SAFE_CALL(cudaGLUnmapBufferObject(vbo));
}



#include "../../src/BulletMultiThreaded/btGpuUtilsSharedCode.h"


