#include "hip/hip_runtime.h"
/*
  FLUIDS v.1 - SPH Fluid Simulator for CPU and GPU
  Copyright (C) 2008. Rama Hoetzlein, http://www.rchoetzlein.com

  ZLib license
  This software is provided 'as-is', without any express or implied
  warranty.  In no event will the authors be held liable for any damages
  arising from the use of this software.

  Permission is granted to anyone to use this software for any purpose,
  including commercial applications, and to alter it and redistribute it
  freely, subject to the following restrictions:

  1. The origin of this software must not be misrepresented; you must not
     claim that you wrote the original software. If you use this software
     in a product, an acknowledgment in the product documentation would be
     appreciated but is not required.
  2. Altered source versions must be plainly marked as such, and must not be
     misrepresented as being the original software.
  3. This notice may not be removed or altered from any source distribution.
*/



#ifndef _PARTICLES_KERNEL_H_
	#define _PARTICLES_KERNEL_H_

	#include <stdio.h>
	#include <math.h>

	#include "fluid_system_host.cuh"

	#define TOTAL_THREADS		65536
	#define BLOCK_THREADS		256
	#define MAX_NBR				80
	
	__constant__	FluidParams		simData;		// simulation data (on device)
	
	__device__ int				bufNeighbor[ TOTAL_THREADS*MAX_NBR ];
	__device__ float			bufNdist[ TOTAL_THREADS*MAX_NBR ];	

	#define COLOR(r,g,b)	( (uint((r)*255.0f)<<24) | (uint((g)*255.0f)<<16) | (uint((b)*255.0f)<<8) )
	#define COLORA(r,g,b,a)	( (uint((r)*255.0f)<<24) | (uint((g)*255.0f)<<16) | (uint((b)*255.0f)<<8) | uint((a)*255.0f) )
	
	#define NULL_HASH		333333
	
	#define OFFSET_CLR		12
	#define OFFSET_NEXT		16
	#define OFFSET_VEL		20
	#define OFFSET_VEVAL	32
	#define OFFSET_PRESS	48
	#define OFFSET_DENS		52
	#define OFFSET_FORCE	56
	

	__global__ void hashParticles ( char* bufPnts, uint2* bufHash, int numPnt )
	{			
		uint ndx = __mul24(blockIdx.x, blockDim.x) + threadIdx.x;	// particle index
		float3* pos = (float3*) (bufPnts + __mul24(ndx, simData.stride) );
		int gz = (pos->z - simData.min.z) * simData.delta.z ;
		int gy = (pos->y - simData.min.y) * simData.delta.y ;
		int gx = (pos->x - simData.min.x) * simData.delta.x ;
		if ( ndx >= numPnt || gx < 0 || gz > simData.res.x-1 || gy < 0 || gy > simData.res.y-1 || gz < 0 || gz > simData.res.z-1 ) 
			bufHash[ndx] = make_uint2( NULL_HASH, ndx );
		else
			bufHash[ndx] = make_uint2( __mul24(__mul24(gz, (int) simData.res.y)+gy, (int) simData.res.x) + gx, ndx );		
		
		__syncthreads ();
	}
	
	__global__ void insertParticles ( char* bufPnts, uint2* bufHash, int* bufGrid, int numPnt, int numGrid )
	{
		uint grid_ndx = __mul24(blockIdx.x, blockDim.x) + threadIdx.x;	// grid cell index		
		
		bufPnts += OFFSET_NEXT;
		bufGrid[grid_ndx] = -1;
		for (int n=0; n < numPnt; n++) {
			if ( bufHash[n].x == grid_ndx ) {
				*(int*) (bufPnts + __mul24(bufHash[n].y, simData.stride)) = bufGrid[grid_ndx];
				bufGrid[grid_ndx] = bufHash[n].y;
			}
		}		
		__syncthreads ();
	}
	
	__global__ void insertParticlesRadix ( char* bufPnts, uint2* bufHash, int* bufGrid, char* bufPntSort, int numPnt, int numGrid )
	{
		uint ndx = __mul24(blockIdx.x, blockDim.x) + threadIdx.x;		// particle index		
		
		uint2 bufHashSort = bufHash[ndx];
		
		__shared__ uint sharedHash[257];	
		sharedHash[threadIdx.x+1] = bufHashSort.x;
		if ( ndx > 0 && threadIdx.x == 0 ) {
			volatile uint2 prevData = bufHash[ndx-1];
			sharedHash[0]  = prevData.x;
		}
		__syncthreads ();
		
		if ( (ndx == 0 || bufHashSort.x != sharedHash[threadIdx.x]) && bufHashSort.x != NULL_HASH ) {
			bufGrid [ bufHashSort.x ] = ndx;			
		}
		if ( ndx < numPnt ) {
			char* src = bufPnts + __mul24( bufHashSort.y, simData.stride );
			char* dest = bufPntSort + __mul24( ndx, simData.stride );
			
			*(float3*)(dest)				= *(float3*)(src);
			*(uint*)  (dest + OFFSET_CLR)	= *(uint*)  (src + OFFSET_CLR);
			*(float3*)(dest + OFFSET_VEL)	= *(float3*)(src + OFFSET_VEL);
			*(float3*)(dest + OFFSET_VEVAL)	= *(float3*)(src + OFFSET_VEVAL);				
			
			*(float*) (dest + OFFSET_DENS)	= 0.0;
			*(float*) (dest + OFFSET_PRESS)	= 0.0;				
			*(float3*) (dest + OFFSET_FORCE)= make_float3(0,0,0);		
			*(int*)   (dest + OFFSET_NEXT)	= bufHashSort.x;			
		} 
		
		__syncthreads ();
		
	}
	
	//__shared__ int ncount [ BLOCK_THREADS ];
	
	__device__ float contributePressure ( int pndx, float3* p, int qndx, int grid_ndx, char* bufPnts, uint2* bufHash )
	{			
		float3* qpos;		
		float3 dist;
		float dsq, c, sum;
		float d = simData.sim_scale;				
		int nbr = __mul24(pndx, MAX_NBR);
						
		sum = 0.0;		
		for ( ; qndx < simData.pnts; qndx++ ) {
			
			if ( bufHash[qndx].x != grid_ndx || qndx == NULL_HASH) break;
			
			if ( qndx != pndx ) {
				qpos = (float3*) ( bufPnts + __mul24(qndx, simData.stride ));	
					
				dist.x = ( p->x - qpos->x )*d;		// dist in cm
				dist.y = ( p->y - qpos->y )*d;
				dist.z = ( p->z - qpos->z )*d;			
				dsq = (dist.x*dist.x + dist.y*dist.y + dist.z*dist.z);			
				if ( dsq < simData.r2 ) {
					c = simData.r2 - dsq;
					sum += c * c * c;				
					if  ( bufNeighbor[nbr] < MAX_NBR ) {
						bufNeighbor[ nbr+bufNeighbor[nbr] ] = qndx;
						bufNdist[ nbr+bufNeighbor[nbr] ] = sqrt(dsq);
						bufNeighbor[nbr]++;
					}
				}				
			}
			//curr = *(int*) (bufPnts + __mul24(curr, simData.stride) + OFFSET_NEXT);
		}		
		return sum;
	}
	
		/*if  ( ncount[threadIdx.x]  < MAX_NBR ) {
				bufNeighbor [ nbr + ncount[threadIdx.x]  ] = curr;
				bufNdist [ nbr + ncount[threadIdx.x]  ] = sqrt(dsq);
				ncount[threadIdx.x]++;
		}*/	
		
	__global__ void computePressure ( char* bufPntSort, int* bufGrid, uint2* bufHash, int numPnt )
	{
		uint ndx = __mul24(blockIdx.x, blockDim.x) + threadIdx.x;	// particle index		

		//if ( ndx < 1024 ) {
		
		float3* pos = (float3*) (bufPntSort + __mul24(ndx, simData.stride));

		// Find 2x2x2 grid cells
		// - Use registers only, no arrays (local-memory too slow)
		int3 cell;
		int gc0, gc1, gc2, gc3, gc4, gc5, gc6, gc7;					
		float gs = simData.smooth_rad / simData.sim_scale;		

		cell.x = max(0, (int)((-gs + pos->x - simData.min.x) * simData.delta.x));
		cell.y = max(0, (int)((-gs + pos->y - simData.min.y) * simData.delta.y));
		cell.z = max(0, (int)((-gs + pos->z - simData.min.z) * simData.delta.z));		
		gc0 = __mul24(__mul24(cell.z, simData.res.y) + cell.y, simData.res.x) + cell.x;
		gc1 = gc0 + 1;
		gc2 = gc0 + simData.res.x;
		gc3 = gc2 + 1;
		if ( cell.z+1 < simData.res.z ) {
			gc4 = gc0 + __mul24(simData.res.x, simData.res.y);
			gc5 = gc4 + 1;
			gc6 = gc4 + simData.res.x;
			gc7 = gc6 + 1;
		}
		if ( cell.x+1 >= simData.res.x ) {
			gc1 = -1; gc3 = -1;
			gc5 = -1; gc7 = -1;
		}
		if ( cell.y+1 >= simData.res.y ) {
			gc2 = -1; gc3 = -1;
			gc6 = -1; gc7 = -1;
		}
		// Sum Pressure
		float sum = 0.0;		
		bufNeighbor[ __mul24(ndx, MAX_NBR) ] = 1;
		if (gc0 != -1 ) sum += contributePressure ( ndx, pos, bufGrid[gc0], gc0, bufPntSort, bufHash );
		if (gc1 != -1 ) sum += contributePressure ( ndx, pos, bufGrid[gc1], gc1, bufPntSort, bufHash );		
		if (gc2 != -1 ) sum += contributePressure ( ndx, pos, bufGrid[gc2], gc2, bufPntSort, bufHash );		
		if (gc3 != -1 ) sum += contributePressure ( ndx, pos, bufGrid[gc3], gc3, bufPntSort, bufHash );	
		if (gc4 != -1 ) sum += contributePressure ( ndx, pos, bufGrid[gc4], gc4, bufPntSort, bufHash );
		if (gc5 != -1 ) sum += contributePressure ( ndx, pos, bufGrid[gc5], gc5, bufPntSort, bufHash );		
		if (gc6 != -1 ) sum += contributePressure ( ndx, pos, bufGrid[gc6], gc6, bufPntSort, bufHash );
		if (gc7 != -1 ) sum += contributePressure ( ndx, pos, bufGrid[gc7], gc7, bufPntSort, bufHash );
		
		// Compute Density & Pressure
		sum = sum * simData.pmass * simData.poly6kern;
		if ( sum == 0.0 ) sum = 1.0;
		*(float*) ((char*)pos + OFFSET_PRESS) = ( sum - simData.rest_dens ) * simData.stiffness;
		*(float*) ((char*)pos + OFFSET_DENS) = 1.0f / sum;			
		
		//}		
		//__syncthreads ();
	}

	__device__ void contributeForce ( float3& force, int pndx, float3* p, int qndx, int grid_ndx, char* bufPnts, uint2* bufHash )
	{
		float press = *(float*) ((char*)p + OFFSET_PRESS);
		float dens = *(float*) ((char*)p + OFFSET_DENS);
		float3 veval = *(float3*) ((char*)p + OFFSET_VEVAL );
		float3 qeval, dist;				
		float c, ndistj, dsq;
		float pterm, dterm, vterm;		
		float3* qpos;				
		float d = simData.sim_scale;				
		
		vterm = simData.lapkern * simData.visc;		
						
		for ( ; qndx < simData.pnts; qndx++ ) {
			
			if ( bufHash[qndx].x != grid_ndx || qndx == NULL_HASH) break;
			
			if ( qndx != pndx ) {
				qpos = (float3*) ( bufPnts + __mul24(qndx, simData.stride ));	
					
				dist.x = ( p->x - qpos->x )*d;		// dist in cm
				dist.y = ( p->y - qpos->y )*d;
				dist.z = ( p->z - qpos->z )*d;			
				dsq = (dist.x*dist.x + dist.y*dist.y + dist.z*dist.z);			
				if ( dsq < simData.r2 ) {				
					ndistj = sqrt(dsq);
					c = ( simData.smooth_rad - ndistj ); 
					dist.x = ( p->x - qpos->x )*d;		// dist in cm
					dist.y = ( p->y - qpos->y )*d;
					dist.z = ( p->z - qpos->z )*d;			
					pterm = -0.5f * c * simData.spikykern * ( press + *(float*)((char*)qpos+OFFSET_PRESS) ) / ndistj;
					dterm = c * dens * *(float*)((char*)qpos+OFFSET_DENS);	
					qeval = *(float3*)((char*)qpos+OFFSET_VEVAL);
					force.x += ( pterm * dist.x + vterm * ( qeval.x - veval.x )) * dterm;
					force.y += ( pterm * dist.y + vterm * ( qeval.y - veval.y )) * dterm;
					force.z += ( pterm * dist.z + vterm * ( qeval.z - veval.z )) * dterm;							
				}
			}
		}				
	}
	
	
	
	__global__ void computeForce ( char* bufPntSort, int* bufGrid, uint2* bufHash, int numPnt )
	{
		uint ndx = __mul24(blockIdx.x, blockDim.x) + threadIdx.x;	// particle index		
		
		//if ( ndx < numPnt ) {
		
		float3* pos = (float3*) (bufPntSort + __mul24(ndx, simData.stride));				
		
		// Find 2x2x2 grid cells
		// - Use registers only, no arrays (local-memory too slow)
		int3 cell;
		int gc0, gc1, gc2, gc3, gc4, gc5, gc6, gc7;					
		float gs = simData.smooth_rad / simData.sim_scale;		

		cell.x = max(0, (int)((-gs + pos->x - simData.min.x) * simData.delta.x));
		cell.y = max(0, (int)((-gs + pos->y - simData.min.y) * simData.delta.y));
		cell.z = max(0, (int)((-gs + pos->z - simData.min.z) * simData.delta.z));		
		gc0 = __mul24(__mul24(cell.z, simData.res.y) + cell.y, simData.res.x) + cell.x;
		gc1 = gc0 + 1;
		gc2 = gc0 + simData.res.x;
		gc3 = gc2 + 1;
		if ( cell.z+1 < simData.res.z ) {
			gc4 = gc0 + __mul24(simData.res.x, simData.res.y);
			gc5 = gc4 + 1;
			gc6 = gc4 + simData.res.x;
			gc7 = gc6 + 1;
		}
		if ( cell.x+1 >= simData.res.x ) {
			gc1 = -1; gc3 = -1;
			gc5 = -1; gc7 = -1;
		}
		if ( cell.y+1 >= simData.res.y ) {
			gc2 = -1; gc3 = -1;
			gc6 = -1; gc7 = -1;
		}
		// Sum Pressure
		float3 force = make_float3(0,0,0);
		if (gc0 != -1 ) contributeForce ( force, ndx, pos, bufGrid[gc0], gc0, bufPntSort, bufHash );
		if (gc1 != -1 ) contributeForce ( force, ndx, pos, bufGrid[gc1], gc1, bufPntSort, bufHash );		
		if (gc2 != -1 ) contributeForce ( force, ndx, pos, bufGrid[gc2], gc2, bufPntSort, bufHash );		
		if (gc3 != -1 ) contributeForce ( force, ndx, pos, bufGrid[gc3], gc3, bufPntSort, bufHash );	
		if (gc4 != -1 ) contributeForce ( force, ndx, pos, bufGrid[gc4], gc4, bufPntSort, bufHash );
		if (gc5 != -1 ) contributeForce ( force, ndx, pos, bufGrid[gc5], gc5, bufPntSort, bufHash );		
		if (gc6 != -1 ) contributeForce ( force, ndx, pos, bufGrid[gc6], gc6, bufPntSort, bufHash );
		if (gc7 != -1 ) contributeForce ( force, ndx, pos, bufGrid[gc7], gc7, bufPntSort, bufHash );
		
		// Update Force
		*(float3*) ((char*)pos + OFFSET_FORCE ) = force;	
		
		//}
		//__syncthreads ();
	}

	
	__global__ void computeForceNbr ( char* bufPntSort, int numPnt )
	{		
		uint ndx = __mul24(blockIdx.x, blockDim.x) + threadIdx.x;	// particle index		
		
		if ( ndx < numPnt ) {
				
		float3* pos = (float3*) (bufPntSort + __mul24(ndx, simData.stride));			
		
		float3* qpos;
		float press = *(float*) ((char*)pos + OFFSET_PRESS);
		float dens = *(float*) ((char*)pos + OFFSET_DENS);
		float3 veval = *(float3*) ((char*)pos + OFFSET_VEVAL );
		float3 qeval, dist, force;		
		float d = simData.sim_scale;
		float c, ndistj;
		float pterm, dterm, vterm;
		vterm = simData.lapkern * simData.visc;
		int nbr = __mul24(ndx, MAX_NBR);
		
		int ncnt = bufNeighbor[ nbr ];		
		
		force = make_float3(0,0,0);
		for (int j=1; j < ncnt; j++) {		// base 1, n[0] = count
			ndistj = bufNdist[ nbr+j ];
			qpos = (float3*) (bufPntSort + __mul24( bufNeighbor[ nbr+j ], simData.stride) );
			c = ( simData.smooth_rad - ndistj ); 
			dist.x = ( pos->x - qpos->x )*d;		// dist in cm
			dist.y = ( pos->y - qpos->y )*d;
			dist.z = ( pos->z - qpos->z )*d;			
			pterm = -0.5f * c * simData.spikykern * ( press + *(float*)((char*)qpos+OFFSET_PRESS) ) / ndistj;
			dterm = c * dens * *(float*)((char*)qpos+OFFSET_DENS);	
			qeval = *(float3*)((char*)qpos+OFFSET_VEVAL);
			force.x += ( pterm * dist.x + vterm * ( qeval.x - veval.x )) * dterm;
			force.y += ( pterm * dist.y + vterm * ( qeval.y - veval.y )) * dterm;
			force.z += ( pterm * dist.z + vterm * ( qeval.z - veval.z )) * dterm;			
		}
		*(float3*) ((char*)pos + OFFSET_FORCE ) = force;
		
		}	
	
	}
		
	__global__ void advanceParticles ( char* bufPntSort, int numPnt, float dt, float ss )
	{		
		uint ndx = __mul24(blockIdx.x, blockDim.x) + threadIdx.x;	// particle index		
		
		if ( ndx < numPnt ) {
				
			// Get particle vars
			float3* pos = (float3*) (bufPntSort + __mul24(ndx, simData.stride));			
			float3* vel = (float3*) ((char*)pos + OFFSET_VEL );
			float3* vel_eval = (float3*) ((char*)pos + OFFSET_VEVAL );
			float3 accel = *(float3*) ((char*)pos + OFFSET_FORCE );
			float3 vcurr, vnext;			

			// Leapfrog integration						
			accel.x *= 0.00020543;			// NOTE - To do: SPH_PMASS should be passed in			
			accel.y *= 0.00020543;
			accel.z *= 0.00020543;			
			accel.z -= 9.8;	
			
			vcurr = *vel;
			vnext.x = accel.x*dt + vcurr.x;	
			vnext.y = accel.y*dt + vcurr.y;	
			vnext.z = accel.z*dt + vcurr.z;			// v(t+1/2) = v(t-1/2) + a(t) dt			
			
			accel.x = (vcurr.x + vnext.x) * 0.5;		// v(t+1) = [v(t-1/2) + v(t+1/2)] * 0.5		used to compute forces later
			accel.y = (vcurr.y + vnext.y) * 0.5;		// v(t+1) = [v(t-1/2) + v(t+1/2)] * 0.5		used to compute forces later
			accel.z = (vcurr.z + vnext.z) * 0.5;		// v(t+1) = [v(t-1/2) + v(t+1/2)] * 0.5		used to compute forces later
			
			*vel_eval = accel;			
			*vel = vnext;
			
			dt /= simData.sim_scale;
			vnext.x = pos->x + vnext.x*dt;
			vnext.y = pos->y + vnext.y*dt;
			vnext.z = pos->z + vnext.z*dt;
			*pos = vnext;						// p(t+1) = p(t) + v(t+1/2) dt			
		}	
		
		__syncthreads ();	
	}

#endif
