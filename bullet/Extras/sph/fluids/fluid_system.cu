#include "hip/hip_runtime.h"
/*
  FLUIDS v.1 - SPH Fluid Simulator for CPU and GPU
  Copyright (C) 2009. Rama Hoetzlein, http://www.rchoetzlein.com

  ZLib license
  This software is provided 'as-is', without any express or implied
  warranty.  In no event will the authors be held liable for any damages
  arising from the use of this software.

  Permission is granted to anyone to use this software for any purpose,
  including commercial applications, and to alter it and redistribute it
  freely, subject to the following restrictions:

  1. The origin of this software must not be misrepresented; you must not
     claim that you wrote the original software. If you use this software
     in a product, an acknowledgment in the product documentation would be
     appreciated but is not required.
  2. Altered source versions must be plainly marked as such, and must not be
     misrepresented as being the original software.
  3. This notice may not be removed or altered from any source distribution.
*/

#include <cutil.h>
#include <cstdlib>
#include <cstdio>
#include <string.h>

#if defined(__APPLE__) || defined(MACOSX)
	#include <GLUT/glut.h>
#else
	#include <GL/glut.h>
#endif
#include <cuda_gl_interop.h>

#include "fluid_system_kern.cu"

extern "C"
{

// Compute number of blocks to create
int iDivUp (int a, int b) {
    return (a % b != 0) ? (a / b + 1) : (a / b);
}
void computeNumBlocks (int numPnts, int minThreads, int &numBlocks, int &numThreads)
{
    numThreads = min( minThreads, numPnts );
    numBlocks = iDivUp ( numPnts, numThreads );
}


void Grid_InsertParticlesCUDA ( uchar* data, uint stride, uint numPoints )
{
    int numThreads, numBlocks;
    computeNumBlocks (numPoints, 256, numBlocks, numThreads);

	// transfer point data to device
    char* pntData;
	size = numPoints * stride;
	hipMalloc( (void**) &pntData, size);
	hipMemcpy( pntData, data, size, hipMemcpyHostToDevice);    

    // execute the kernel
    insertParticles<<< numBlocks, numThreads >>> ( pntData, stride );
    
    // transfer data back to host
    hipMemcpy( data, pntData, hipMemcpyDeviceToHost);
    
    // check if kernel invocation generated an error
    CUT_CHECK_ERROR("Kernel execution failed");
    CUDA_SAFE_CALL(cudaGLUnmapBufferObject(vboPos));
}