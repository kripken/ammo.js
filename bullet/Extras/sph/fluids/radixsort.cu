#include "hip/hip_runtime.h"
/*
 * Copyright 1993-2006 NVIDIA Corporation.  All rights reserved.
 *
 * NOTICE TO USER:
 *
 * This source code is subject to NVIDIA ownership rights under U.S. and
 * international Copyright laws.
 *
 * NVIDIA MAKES NO REPRESENTATION ABOUT THE SUITABILITY OF THIS SOURCE
 * CODE FOR ANY PURPOSE.  IT IS PROVIDED "AS IS" WITHOUT EXPRESS OR
 * IMPLIED WARRANTY OF ANY KIND.  NVIDIA DISCLAIMS ALL WARRANTIES WITH
 * REGARD TO THIS SOURCE CODE, INCLUDING ALL IMPLIED WARRANTIES OF
 * MERCHANTABILITY, NONINFRINGEMENT, AND FITNESS FOR A PARTICULAR PURPOSE.
 * IN NO EVENT SHALL NVIDIA BE LIABLE FOR ANY SPECIAL, INDIRECT, INCIDENTAL,
 * OR CONSEQUENTIAL DAMAGES, OR ANY DAMAGES WHATSOEVER RESULTING FROM LOSS
 * OF USE, DATA OR PROFITS, WHETHER IN AN ACTION OF CONTRACT, NEGLIGENCE
 * OR OTHER TORTIOUS ACTION, ARISING OUT OF OR IN CONNECTION WITH THE USE
 * OR PERFORMANCE OF THIS SOURCE CODE.
 *
 * U.S. Government End Users.  This source code is a "commercial item" as
 * that term is defined at 48 C.F.R. 2.101 (OCT 1995), consisting  of
 * "commercial computer software" and "commercial computer software
 * documentation" as such terms are used in 48 C.F.R. 12.212 (SEPT 1995)
 * and is provided to the U.S. Government only as a commercial end item.
 * Consistent with 48 C.F.R.12.212 and 48 C.F.R. 227.7202-1 through
 * 227.7202-4 (JUNE 1995), all U.S. Government End Users acquire the
 * source code with only those rights set forth herein.
 */

/* Radixsort project with key/value and arbitrary datset size support
 * which demonstrates the use of CUDA in a multi phase sorting 
 * computation.
 * Host code.
 */

#include "radixsort.cuh"
#include "radixsort_kernel.cu"

extern "C"
{

////////////////////////////////////////////////////////////////////////////////
//! Perform a radix sort
//! Sorting performed in place on passed arrays.
//!
//! @param pData0       input and output array - data will be sorted
//! @param pData1       additional array to allow ping pong computation
//! @param elements     number of elements to sort
////////////////////////////////////////////////////////////////////////////////
void RadixSort(KeyValuePair *pData0, KeyValuePair *pData1, uint elements, uint bits)
{
    // Round element count to total number of threads for efficiency
    uint elements_rounded_to_3072;
    int modval = elements % 3072;
    if( modval == 0 )
        elements_rounded_to_3072 = elements;
    else
        elements_rounded_to_3072 = elements + (3072 - (modval));

    // Iterate over n bytes of y bit word, using each byte to sort the list in turn
    for (uint shift = 0; shift < bits; shift += RADIX)
    {
        // Perform one round of radix sorting

        // Generate per radix group sums radix counts across a radix group
        RadixSum<<<NUM_BLOCKS, NUM_THREADS_PER_BLOCK, GRFSIZE>>>(pData0, elements, elements_rounded_to_3072, shift);
        // Prefix sum in radix groups, and then between groups throughout a block
        RadixPrefixSum<<<PREFIX_NUM_BLOCKS, PREFIX_NUM_THREADS_PER_BLOCK, PREFIX_GRFSIZE>>>();
        // Sum the block offsets and then shuffle data into bins
        RadixAddOffsetsAndShuffle<<<NUM_BLOCKS, NUM_THREADS_PER_BLOCK, SHUFFLE_GRFSIZE>>>(pData0, pData1, elements, elements_rounded_to_3072, shift); 

        // Exchange data pointers
        KeyValuePair* pTemp = pData0;
        pData0 = pData1;
        pData1 = pTemp;
   }
}

}
