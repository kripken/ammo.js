#include "hip/hip_runtime.h"
/*
  FLUIDS v.1 - SPH Fluid Simulator for CPU and GPU
  Copyright (C) 2008. Rama Hoetzlein, http://www.rchoetzlein.com

  ZLib license
  This software is provided 'as-is', without any express or implied
  warranty.  In no event will the authors be held liable for any damages
  arising from the use of this software.

  Permission is granted to anyone to use this software for any purpose,
  including commercial applications, and to alter it and redistribute it
  freely, subject to the following restrictions:

  1. The origin of this software must not be misrepresented; you must not
     claim that you wrote the original software. If you use this software
     in a product, an acknowledgment in the product documentation would be
     appreciated but is not required.
  2. Altered source versions must be plainly marked as such, and must not be
     misrepresented as being the original software.
  3. This notice may not be removed or altered from any source distribution.
*/


//#include "C:\CUDA\common\inc\cutil.h"				// cutil32.lib
#include <string.h>
#include "../CUDA/btCudaDefines.h"



#if defined(__APPLE__) || defined(MACOSX)
	#include <GLUT/glut.h>
#else
	#include <GL/glut.h>
#endif
#include <cuda_gl_interop.h>

#include "radixsort.cu"
#include "fluid_system_kern.cu"			// build kernel

FluidParams					fcuda;

__device__ char*			bufPnts;		// point data (array of Fluid structs)
__device__ char*			bufPntSort;		// point data (array of Fluid structs)
__device__ uint*			bufHash[2];		// point grid hash
__device__ int*				bufGrid;	

	

extern "C"
{
// Initialize CUDA
void cudaInit(int argc, char **argv)
{   
    //CUT_DEVICE_INIT(argc, argv);
 
	hipDeviceProp_t p;
	hipGetDeviceProperties ( &p, 0);
	
	printf ( "-- CUDA --\n" );
	printf ( "Name:       %s\n", p.name );
	printf ( "Revision:   %d.%d\n", p.major, p.minor );
	printf ( "Global Mem: %d\n", p.totalGlobalMem );
	printf ( "Shared/Blk: %d\n", p.sharedMemPerBlock );
	printf ( "Regs/Blk:   %d\n", p.regsPerBlock );
	printf ( "Warp Size:  %d\n", p.warpSize );
	printf ( "Mem Pitch:  %d\n", p.memPitch );
	printf ( "Thrds/Blk:  %d\n", p.maxThreadsPerBlock );
	printf ( "Const Mem:  %d\n", p.totalConstMem );
	printf ( "Clock Rate: %d\n", p.clockRate );	
	
	BT_GPU_SAFE_CALL ( hipMalloc ( (void**) &bufPnts, 10 ) );	
	BT_GPU_SAFE_CALL ( hipMalloc ( (void**) &bufPntSort, 10 ) );
	BT_GPU_SAFE_CALL ( hipMalloc ( (void**) &bufHash, 10 ) );	
	BT_GPU_SAFE_CALL ( hipMalloc ( (void**) &bufGrid, 10 ) );	
};
	
// Compute number of blocks to create
int iDivUp (int a, int b) {
    return (a % b != 0) ? (a / b + 1) : (a / b);
}
void computeNumBlocks (int numPnts, int maxThreads, int &numBlocks, int &numThreads)
{
    numThreads = min( maxThreads, numPnts );
    numBlocks = iDivUp ( numPnts, numThreads );
}

void FluidClearCUDA ()
{
	BT_GPU_SAFE_CALL ( hipFree ( bufPnts ) );	
	BT_GPU_SAFE_CALL ( hipFree ( bufPntSort ) );
	BT_GPU_SAFE_CALL ( hipFree ( bufHash[0] ) );	
	BT_GPU_SAFE_CALL ( hipFree ( bufHash[1] ) );	
	BT_GPU_SAFE_CALL ( hipFree ( bufGrid ) );
}


void FluidSetupCUDA ( int num, int stride, float3 min, float3 max, float3 res, float3 size, int chk )
{	
	fcuda.min = make_float3(min.x, min.y, min.z);
	fcuda.max = make_float3(max.x, max.y, max.z);
	fcuda.res = make_float3(res.x, res.y, res.z);
	fcuda.size = make_float3(size.x, size.y, size.z);	
	fcuda.pnts = num;
	fcuda.delta.x = res.x / size.x;
	fcuda.delta.y = res.y / size.y;
	fcuda.delta.z = res.z / size.z;
	fcuda.cells = res.x*res.y*res.z;
	fcuda.chk = chk;
		
    computeNumBlocks ( fcuda.pnts, 256, fcuda.numBlocks, fcuda.numThreads);			// particles
    computeNumBlocks ( fcuda.cells, 256, fcuda.gridBlocks, fcuda.gridThreads);		// grid cell
    
    fcuda.szPnts = (fcuda.numBlocks * fcuda.numThreads) * stride;        
    fcuda.szHash = (fcuda.numBlocks * fcuda.numThreads) * sizeof(uint2);		// <cell, particle> pairs
    fcuda.szGrid = (fcuda.gridBlocks * fcuda.gridThreads) * sizeof(uint);    
    fcuda.stride = stride;
    printf ( "pnts: %d, t:%dx%d=%d, bufPnts:%d, bufHash:%d\n", fcuda.pnts, fcuda.numBlocks, fcuda.numThreads, fcuda.numBlocks*fcuda.numThreads, fcuda.szPnts, fcuda.szHash );
    printf ( "grds: %d, t:%dx%d=%d, bufGrid:%d, Res: %dx%dx%d\n", fcuda.cells, fcuda.gridBlocks, fcuda.gridThreads, fcuda.gridBlocks*fcuda.gridThreads, fcuda.szGrid, (int) fcuda.res.x, (int) fcuda.res.y, (int) fcuda.res.z );	

	BT_GPU_SAFE_CALL ( hipMalloc ( (void**) &bufPnts, fcuda.szPnts ) );	
	BT_GPU_SAFE_CALL ( hipMalloc ( (void**) &bufPntSort, fcuda.szPnts ) );	
	BT_GPU_SAFE_CALL ( hipMalloc ( (void**) &bufHash[0], fcuda.szHash ) );	
	BT_GPU_SAFE_CALL ( hipMalloc ( (void**) &bufHash[1], fcuda.szHash ) );	
	BT_GPU_SAFE_CALL ( hipMalloc ( (void**) &bufGrid, fcuda.szGrid ) );
	
	printf ( "POINTERS\n");
	printf ( "bufPnts:    %p\n", bufPnts );
	printf ( "bufPntSort: %p\n", bufPntSort );
	printf ( "bufHash0:   %p\n", bufHash[0] );
	printf ( "bufHash1:   %p\n", bufHash[1] );
	printf ( "bufGrid:    %p\n", bufGrid );
	
	BT_GPU_SAFE_CALL ( hipMemcpyToSymbol(HIP_SYMBOL( simData), &fcuda, sizeof(FluidParams) ) );
	hipDeviceSynchronize ();
}

void FluidParamCUDA ( float sim_scale, float smooth_rad, float mass, float rest, float stiff, float visc )
{
	fcuda.sim_scale = sim_scale;
	fcuda.smooth_rad = smooth_rad;
	fcuda.r2 = smooth_rad * smooth_rad;
	fcuda.pmass = mass;
	fcuda.rest_dens = rest;	
	fcuda.stiffness = stiff;
	fcuda.visc = visc;
	
	fcuda.pdist = pow ( fcuda.pmass / fcuda.rest_dens, 1/3.0f );
	fcuda.poly6kern = 315.0f / (64.0f * 3.141592 * pow( smooth_rad, 9.0f) );
	fcuda.spikykern = -45.0f / (3.141592 * pow( smooth_rad, 6.0f) );
	fcuda.lapkern = 45.0f / (3.141592 * pow( smooth_rad, 6.0f) );	

	BT_GPU_SAFE_CALL( hipMemcpyToSymbol(HIP_SYMBOL( simData), &fcuda, sizeof(FluidParams) ) );
	hipDeviceSynchronize ();
}

void TransferToCUDA ( char* data, int* grid, int numPoints )
{
	BT_GPU_SAFE_CALL( hipMemcpy ( bufPnts, data, numPoints * fcuda.stride, hipMemcpyHostToDevice ) );
	hipDeviceSynchronize ();
}

void TransferFromCUDA ( char* data, int* grid, int numPoints )
{
	BT_GPU_SAFE_CALL( hipMemcpy ( data, bufPntSort, numPoints * fcuda.stride, hipMemcpyDeviceToHost ) );	
	hipDeviceSynchronize ();	
	
	BT_GPU_SAFE_CALL( hipMemcpy ( grid, bufGrid, fcuda.cells * sizeof(uint), hipMemcpyDeviceToHost ) );			
}

void Grid_InsertParticlesCUDA ()
{
	BT_GPU_SAFE_CALL( hipMemset ( bufHash[0], 0, fcuda.szHash ) );
	
	hashParticles<<< fcuda.numBlocks, fcuda.numThreads>>> ( bufPnts, (uint2*) bufHash[0], fcuda.pnts );	
	BT_GPU_CHECK_ERROR( "Kernel execution failed");
	hipDeviceSynchronize ();
	
	//int buf[20000];		
	/*printf ( "HASH: %d (%d)\n", fcuda.pnts, fcuda.numBlocks*fcuda.numThreads );
	BT_GPU_SAFE_CALL( hipMemcpy ( buf, bufHash[0], fcuda.pnts * 2*sizeof(uint), hipMemcpyDeviceToHost ) );		
	//for (int n=0; n < fcuda.numBlocks*fcuda.numThreads; n++) {		
	for (int n=0; n < 100; n++) {
		printf ( "%d: <%d,%d>\n", n, buf[n*2], buf[n*2+1] );
	}*/
	 
	RadixSort( (KeyValuePair *) bufHash[0], (KeyValuePair *) bufHash[1], fcuda.pnts, 32);
	BT_GPU_CHECK_ERROR( "Kernel execution failed");
	hipDeviceSynchronize ();
	
	/*printf ( "HASH: %d (%d)\n", fcuda.pnts, fcuda.numBlocks*fcuda.numThreads );
	BT_GPU_SAFE_CALL( hipMemcpy ( buf, bufHash[0], fcuda.pnts * 2*sizeof(uint), hipMemcpyDeviceToHost ) );		
	//for (int n=0; n < fcuda.numBlocks*fcuda.numThreads; n++) {		
	for (int n=0; n < 100; n++) {
		printf ( "%d: <%d,%d>\n", n, buf[n*2], buf[n*2+1] );
	}*/
	
	// insertParticles<<< fcuda.gridBlocks, fcuda.gridThreads>>> ( bufPnts, (uint2*) bufHash[0], bufGrid, fcuda.pnts, fcuda.cells );			
	
	BT_GPU_SAFE_CALL( hipMemset ( bufGrid, NULL_HASH, fcuda.cells * sizeof(uint) ) );
	
	insertParticlesRadix<<< fcuda.numBlocks, fcuda.numThreads>>> ( bufPnts, (uint2*) bufHash[0], bufGrid, bufPntSort, fcuda.pnts, fcuda.cells );
	BT_GPU_CHECK_ERROR( "Kernel execution failed");
	hipDeviceSynchronize ();	
    
    /*printf ( "GRID: %d\n", fcuda.cells );
	BT_GPU_SAFE_CALL( hipMemcpy ( buf, bufGrid, fcuda.cells * sizeof(uint), hipMemcpyDeviceToHost ) );		
	*for (int n=0; n < 100; n++) {		
		printf ( "%d: %d\n", n, buf[n]);
	}*/
}

void SPH_ComputePressureCUDA ()
{
	computePressure<<< fcuda.numBlocks, fcuda.numThreads>>> ( bufPntSort, bufGrid, (uint2*) bufHash[0], fcuda.pnts );	
    BT_GPU_CHECK_ERROR( "Kernel execution failed");
    hipDeviceSynchronize ();	
}

void SPH_ComputeForceCUDA ()
{
	//-- standard force
	//computeForce<<< fcuda.numBlocks, fcuda.numThreads>>> ( bufPntSort, bufGrid, (uint2*) bufHash[0], fcuda.pnts );	
	
	// Force using neighbor table
	computeForceNbr<<< fcuda.numBlocks, fcuda.numThreads>>> ( bufPntSort, fcuda.pnts );	
    BT_GPU_CHECK_ERROR( "Kernel execution failed");
    hipDeviceSynchronize ();	
}

void SPH_AdvanceCUDA ( float dt, float ss )
{
	advanceParticles<<< fcuda.numBlocks, fcuda.numThreads>>> ( bufPntSort, fcuda.pnts, dt, ss );
    BT_GPU_CHECK_ERROR( "Kernel execution failed");
    hipDeviceSynchronize ();
}

}	// extern C




   	//----------- Per frame: Malloc/Free, Host<->Device
	// transfer point data to device    
    /*char* pntData;
	int size = (fcuda.numBlocks*fcuda.numThreads) * stride;
	hipMalloc( (void**) &pntData, size);
	hipMemcpy( pntData, data, numPoints*stride, hipMemcpyHostToDevice);  	
    insertParticles<<< fcuda.numBlocks, fcuda.numThreads >>> ( pntData, stride, numPoints );
    hipMemcpy( data, pntData, numPoints*stride, hipMemcpyDeviceToHost);    
    hipFree( pntData );*/
